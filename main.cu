#include <hip/hip_runtime.h>

#include <cstdint>
#include <cstdio>
#include <cuda_device_runtime_api.h>
// mixed types work
__global__ void addTest(const int* bcdasdk, int* c) {
  printf("hello from addTest()\n");
}

// twice the same type in a parameter does not work
__global__ void addTest2(void* a, void* b) {
  printf("hello from addTest2()\n");
}

template<typename... Args>
__global__ void execFnPtr(void(*f)(Args...)){
    (*f)<<<1,1>>>(nullptr,nullptr);
}


template <typename F, F f>
__device__ F deviceSymbol = f;

int main() {
  void (*kernelFuncPtr)(const int*, int*);
  auto err = hipMemcpyFromSymbol(&kernelFuncPtr,
                       HIP_SYMBOL(deviceSymbol<decltype(&addTest), &addTest>),
                       sizeof(void*));
  printf("this pointer on the device is: %p\n", kernelFuncPtr);
  execFnPtr<<<1, 1>>>(kernelFuncPtr);

  // This code does not compile on GCC7/CUDA10, comment it out and it should work
  void (*kernelFuncPtr2)(void*, void*);
  err = hipMemcpyFromSymbol(&kernelFuncPtr2,
                       HIP_SYMBOL(deviceSymbol<decltype(&addTest2), &addTest2>),
                       sizeof(void*));
  printf("this pointer will cause a compile-error: %p\n", kernelFuncPtr2);

  execFnPtr<<<1, 1>>>(kernelFuncPtr2);

  return 0;
}
