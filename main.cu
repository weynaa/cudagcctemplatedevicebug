#include <iostream>
#include <cstdint>
#include <hip/hip_runtime.h>

//mixed types work
__global__
void addTest( const int* bcdasdk, int* c){

}

//twice the same type in a parameter does not work
__global__
void addTest2( void* a,void*b){

}


template <typename F, F f>
__device__ F deviceSymbol = f;

int main() {

    void* kernelFuncPtr;
    hipMemcpyFromSymbol(&kernelFuncPtr,HIP_SYMBOL(&deviceSymbol<decltype(&addTest),&addTest>),sizeof(void*));
    printf("this pointer on the device is: %p",kernelFuncPtr);
    //This code does not compile on GCC7/CUDA10
    void* kernelFuncPtr2;
    hipMemcpyFromSymbol(&kernelFuncPtr2,HIP_SYMBOL(&deviceSymbol<decltype(&addTest2),&addTest2>),sizeof(void*));
    printf("this pointer will cause a compile-error: %p",kernelFuncPtr2);

    return 0;
}
